#include "hip/hip_runtime.h"
#ifndef __MATRIXCUDA_H__
#define __MATRIXCUDA_H__
#include "clad/Differentiator/Differentiator.h"
#include <assert.h>
#include <iostream>
#include <math.h>
#include <vector>
using namespace std;

// define network activation; can be Liniar/Step/Ramp/Hyperbolic/Tangent,
// Gaussian Function, here
inline __host__ __device__ double activationFunc(double x) {
  return (1.0 / (1.0 + exp(-x)));
}

// determine the partial derivatives with respect to the inputs for the
// activation function and loss

inline __host__ __device__ void
activationFunc_grad(double x, clad::array_ref<double> _result);
// inline auto af_backpropagate = clad::gradient(activationFunc);

inline __host__ __device__ double neuronOutput(double *weight, double *input,
                                               int input_dim) {
  double neuron_output = 0.0;
  for (int i = 0; i < input_dim; i++) {
    neuron_output += weight[i] * input[i];
  }
  neuron_output += weight[input_dim]; // account for the bias
  return neuron_output;
}

// most famous cost functions are: Quadratic Cost (Root Mean Square), Cross
// Entropy, Exponential
inline __host__ __device__ double lossFunc(double expected_output,
                                           double *weights, double *input,
                                           int input_dim) {
  double neuron_output = activationFunc(
      neuronOutput(weights, input, input_dim)); // account for the bias

  double loss = -1 * (expected_output * log(neuron_output)) +
                (1 - expected_output) * log(1 - neuron_output);
  return loss;
}

__inline__ __host__ __device__ double
lossFunc_grad_1_2(double expected_output, double *weights, double *input, int input_dim, clad::array_ref<double> network_backprop,
                  clad::array_ref<double> input_backprop);

// inline auto error_backpropagate = clad::gradient(lossFunc, "weights,
// input");

class Network {
private:
  // nr of input/output sneurons
  std::size_t input_dim, output_dim, sizeArray;
  double lr; // learning rate
  double *output_values;
  double *input_values;
  bool are_weights_allocated = false;

public:
  double *weights;
  Network()
      : input_dim(2), output_dim(1), sizeArray((input_dim + 1) * output_dim),
        lr(0.15) {
    hipError_t err = hipMallocManaged(&weights, sizeArray * sizeof(double));
    if (err != hipSuccess)
      printf("1-Memory allocation failed: %d\n", err);
    are_weights_allocated = true;
  }

  Network(const size_t input_neurons, const size_t output_neurons,
          double learning_rate)
      : input_dim(input_neurons), output_dim(output_neurons),
        sizeArray((input_dim + 1) * output_dim), lr(learning_rate) {
    hipError_t err = hipMallocManaged(&weights, sizeArray * sizeof(double));
    if (err != hipSuccess)
      printf("2-Memory allocation failed: %d\n", err);
    //    srand(time(0));
    size_t i, j, l;
    for (i = 0; i < output_dim; i++) {
      // the extra weight needs to account for the bias
      for (j = 0; j < input_dim + 1; j++) {
        l = i * (input_dim + 1) + j;
        weights[l] = ((double)rand() / RAND_MAX);
      }
    }
    are_weights_allocated = true;
  }

  Network(size_t input_neurons, size_t output_neurons, double learning_rate,
          double *trained_weights)
      : input_dim(input_neurons), output_dim(output_neurons),
        sizeArray((input_dim + 1) * output_dim), lr(learning_rate) {
    weights = trained_weights;
    are_weights_allocated = false;
  }

  ~Network() {
    if (are_weights_allocated)
      hipFree(weights);
  }

  __host__ __device__ double *feedForward(double *input, int input_size) {
    assert(input_size == input_dim);
    output_values = new double[output_dim];
    // double input_vals[input_values.size()];
    // std::copy(input_values.begin(), input_values.end(), input_vals);

    size_t i, j, l;
    for (i = 0; i < output_dim; i++) {
      // the extra weight needs to account for the bias
      double *weight = new double[input_dim + 1];
      for (j = 0; j < input_dim + 1; j++) {
        l = i * (input_dim + 1) + j;
        weight[j] = weights[l];
      }
      output_values[i] = activationFunc(neuronOutput(weight, input, input_dim));
    }
    return output_values;
  }

  __host__ __device__ void backPropagate(double *input, int inp_dim,
                                         double *exp_values, int exp_dim) {
    assert(inp_dim == input_dim);
    assert(exp_dim == output_dim);
    // use the direction of the steepest descent, learning rate and activation
    // function to update $
    size_t out, j, l;
    for (out = 0; out < exp_dim; out++) {
      double activation_backprop[1] = {};
      activationFunc_grad(output_values[out], activation_backprop);

      double *local_weights = new double[input_dim + 1];
      for (j = 0; j < input_dim + 1; j++) {
        l = out * (input_dim + 1) + j;
        local_weights[j] = weights[l];
      }

      double *network_backprop = new double[input_dim + 1]();
      double *input_backprop = new double[input_dim]();
      //
      lossFunc_grad_1_2(
          exp_values[out], local_weights, input, input_dim,
          clad::array_ref<double>(network_backprop, input_dim + 1),
          clad::array_ref<double>(input_backprop, input_dim));

      for (int w = 0; w < input_dim + 1; w++) {
        weights[out * (input_dim + 1) + w] -=
            (lr * network_backprop[w] * activation_backprop[0]);
      }
      //      printf("i: %g %g; w: %g %g %g\n", input[0], input[1],
      //      network_backprop[0], network_backprop[1], network_backprop[2]);
    }
  }

  __host__ __device__ double *train(double *train_data, int data_sample_dim) {
    assert(data_sample_dim == input_dim + output_dim);
    feedForward(train_data, input_dim);
    backPropagate(train_data, input_dim, train_data + input_dim, output_dim);
    return weights;
  }

  __host__ __device__ void test(double *train_data, int nr_samples,
                                int sample_input_dim) {
    assert(sample_input_dim == input_dim + output_dim);
    // printf("prior weights: %f \n", weights[0]);
    // printf("prior weights: %f \n", trained_weights[0]);
    // assert(sizeof(weights) == sizeof(trained_weights));
    // weights = trained_weights;
    printf("after weights: %f \n", weights[0]);
    double acc = 0.0;
    for (int i = 0; i < nr_samples; i++) {
      feedForward(train_data + i * sample_input_dim, input_dim);
      if (output_dim == 1) {
        if (output_values[0] > 0.5 &&
            train_data[i * sample_input_dim + input_dim] == 1.0)
          acc++;
        else if (output_values[0] < 0.5 &&
                 train_data[i * sample_input_dim + input_dim] == 0.0)
          acc++;
        printf("%0.6g\n", output_values[0]);
      } else {
        int max_output_idx = 0, max_data_idx = 0;
        for (int j = 1; j < output_dim; j++) {
          if (output_values[max_output_idx] < output_values[j])
            max_output_idx = j;
          if (train_data[i * sample_input_dim + input_dim + max_data_idx] <
              train_data[i * sample_input_dim + input_dim + j])
            max_data_idx = j;
        }
        acc += (max_output_idx == max_data_idx);
      }
    }
    printf("\nAccuracy: %f \n", acc / nr_samples);
  }

  // __host__ __device__ void assignValue(size_t l, double value){
  //     double activation_backprop[1] = {};
  //     activationFunc_grad(2.0, activation_backprop);
  //     // af_backpropagate.execute(2.0, activation_backprop);

  //     // weights[l] = activation_backprop[0];
  //     weights[l] = value;

  // }

  __host__ __device__ double *returnWeights() { return weights; }

  __host__ __device__ void displayArray() {
    size_t i, j, l;
    for (i = 0; i < (input_dim + 1); ++i) {
      for (j = 0; j < output_dim; ++j) {
        l = i * output_dim + j;
        printf("%f\t", weights[l]);
      }
      // cout<<endl;
      printf("\n");
    }
  }
};
#endif